#include "hip/hip_runtime.h"
#include "solver_nonlinear_full.cuh"

#include <vector>
#include <iostream>

#include "utilities.hpp"


namespace gmf {

/*
 * We need to restrict solution v and an adjusted version of the RHS
 * of the equation.
 * 
 * If R is some restriction operator, then the adjusted coarsened RHS is
 * RHS <- R(RHS) + R(A(v)) - A(R(v))
 */
void SolverNonlinearFull::restrict(const int lvl) {
    assert(lvl > 0);
    Array& v = m_levels[lvl].solution;
    Array& Av = m_levels[lvl].temporary;
    Array& rhs = m_levels[lvl].rhs;
    Grid& grid = m_levels[lvl].grid;

    Array& coarse_v = m_levels[lvl-1].solution;
    Array& coarse_Av = m_levels[lvl-1].temporary;
    Array& coarse_rhs = m_levels[lvl-1].rhs;
    Grid& coarse_grid = m_levels[lvl-1].grid;

    if (m_levels[lvl].on_gpu) {
        m_lhs->run_device(Av, v, *m_boundary_conditions, grid);

        m_restrictor->run_device(v, coarse_v, *m_boundary_conditions);
        m_restrictor->run_device(Av, coarse_Av, *m_boundary_conditions);
        m_restrictor->run_device(rhs, coarse_rhs, *m_boundary_conditions);

        const int threadsPerBlock = std::min(m_max_threads_per_block, coarse_rhs.size() - 1);
        const int blocksPerGrid = (coarse_rhs.size() + threadsPerBlock - 1) / threadsPerBlock;
        kernel_sub<<<blocksPerGrid, threadsPerBlock>>>(coarse_rhs, coarse_Av, coarse_rhs);

        // Recalculate Av with the coarse v, then add to RHS
        m_lhs->run_device(coarse_Av, coarse_v, *m_boundary_conditions, coarse_grid);
        kernel_add<<<blocksPerGrid, threadsPerBlock>>>(coarse_rhs, coarse_Av, coarse_rhs);

        // Store coarse solution temporarily to calculate error AFTER
        // relaxing/solving on the coarser mesh.
        // To save memory, just store this in the FINE temporary array
        Array& temporary = m_levels[lvl].temporary;
        kernel_copy<<<blocksPerGrid, threadsPerBlock>>>(coarse_v, temporary, coarse_v.size());

        if (!m_levels[lvl-1].on_gpu)
            cudaCheck(hipDeviceSynchronize());

    } else {
        m_lhs->run_host(Av, v, *m_boundary_conditions, grid);

        m_restrictor->run_host(v, coarse_v, *m_boundary_conditions);
        m_restrictor->run_host(Av, coarse_Av, *m_boundary_conditions);
        m_restrictor->run_host(rhs, coarse_rhs, *m_boundary_conditions);

        // Subtract coarsened LHS calculated with the fine solution
        for (int i = 0; i < coarse_rhs.size(); ++i)
            coarse_rhs[i] -= coarse_Av[i];

        // Recalculate Av with the coarse v, then add to RHS
        m_lhs->run_host(coarse_Av, coarse_v, *m_boundary_conditions, coarse_grid);
        for (int i = 0; i < coarse_rhs.size(); ++i) {
            coarse_rhs[i] += coarse_Av[i];
        }

        // Store coarse solution temporarily to calculate error AFTER
        // relaxing/solving on the coarser mesh.
        // To save memory, just store this in the FINE temporary array
        Array& temporary = m_levels[lvl].temporary;
        for (int i = 0; i < coarse_v.size(); ++i)
            temporary[i] = coarse_v[i];
    }
}

/*
 * Interpolate errors, NOT solutions.
 *
 * Steps are the following:
 * 1) Calculate the coarse error e = v - v0
 * 2) Interpolate the coarse error to the fine grid
 * 3) Correct the fine solution with the interopolated error
 *    v = v0 + I(e)
 */
void SolverNonlinearFull::correct(const int lvl) {
    assert(lvl > 0);
    Array& solution_prev = m_levels[lvl].temporary; // NB: prev coarse solution intentionally stored in fine temporary array
    Array& solution = m_levels[lvl-1].solution;
    Array& error = m_levels[lvl-1].temporary;

    Array& fine_error = m_levels[lvl].temporary;
    Array& fine_solution = m_levels[lvl].solution;

    if (m_levels[lvl].on_gpu) {
        // Calculate error
        int threadsPerBlock = std::min(m_max_threads_per_block, error.size() - 1);
        int blocksPerGrid = (error.size() + threadsPerBlock - 1) / threadsPerBlock;
        kernel_sub<<<blocksPerGrid, threadsPerBlock>>>(solution, solution_prev, error);

        m_interpolator->run_device(error, fine_error, *m_boundary_conditions);

        // Correct the solution with the error
        threadsPerBlock = std::min(m_max_threads_per_block, fine_solution.size() - 1);
        blocksPerGrid = (fine_solution.size() + threadsPerBlock - 1) / threadsPerBlock;
        kernel_add<<<blocksPerGrid, threadsPerBlock>>>(fine_solution, fine_error, fine_solution);
    } else {
        // Calculate error
        for (int i = 0; i < error.size(); ++i)
            error[i] = solution[i] - solution_prev[i];

        m_interpolator->run_host(error, fine_error, *m_boundary_conditions);

        // Correct the solution with the error
        for (int i = 0; i < fine_solution.size(); ++i)
            fine_solution[i] += fine_error[i];
    }
}

} // namespace gmf
