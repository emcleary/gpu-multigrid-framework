#include "hip/hip_runtime.h"
#include "norm_amax.cuh"

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "src/array.hpp"
#include "src/grid.hpp"
#include "src/utilities.hpp"

namespace gmf {
namespace modules {

NormAmax::NormAmax() {
    cublasCheck(hipblasCreate(&m_handle));
}

NormAmax::~NormAmax() {
    cublasCheck(hipblasDestroy(m_handle));
}

double NormAmax::run_host(const Array& array, const Grid& grid) {
    double norm = 0;
    for (int i = 0; i < array.size(); ++i)
        norm = std::max(norm, std::abs(array[i]));
    return norm;
}

double NormAmax::run_device(const Array& array, const Grid& grid) {
    int index;
    cublasCheck(hipblasIdamax(m_handle, array.size(), array.data(), 1, &index));
    return std::abs(array[index - 1]);
}

} // namespace modules
} // namespace gmf
