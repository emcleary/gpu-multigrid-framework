#include "hip/hip_runtime.h"
#include "iterator_async.cuh"

#include <hip/hip_runtime.h>

#include "schemes.cuh"
#include "src/array.hpp"

using gmf::Array;
using gmf::ArrayRaw;
using gmf::Grid;
using gmf::modules::BoundaryConditions;


namespace async {
__global__
void kernel(ArrayRaw v, const ArrayRaw f, const double h2) {
    static const int n_colors = 2;
    int index = n_colors * (threadIdx.x + blockDim.x * blockIdx.x);
    const int stride = n_colors * blockDim.x * gridDim.x;

    for (int color = 1; color <= n_colors; ++color) {
        int idx = index + color;
        while (idx < v.size()) {
            if (0 < idx && idx < v.size() - 1)
                v[idx] = eval_gs(v, f, h2, idx);
            idx += stride;
        }
        __syncthreads();
    }
}

__global__
void boundaries(ArrayRaw v, const ArrayRaw f, const BoundaryConditions bcs, const double h2) {
    const int n = v.size() - 1;
    
    if (bcs.is_periodic()) {
        v[0] = (v[n-1] + v[1] + f[0] * h2) / 2;
        v[n] = v[0];
    } else {
        if (bcs.is_left_dirichlet()) {
            v[0] = f[0];
        } else { // neumann
            v[0] = v[1] + f[0] * h2 / 2;
        }

        if (bcs.is_right_dirichlet()) {
            v[n] = f[n];
        } else { // neumann
            v[n] = v[n-1] + f[n] * h2 / 2;
        }
    }
}
} // namespace async


void IteratorAsync::run_device(Array& v, const Array& f, const BoundaryConditions& bcs, const Grid& grid) {
    const double h2 = grid.get_cell_width() * grid.get_cell_width();
    size_t i = 1 << (int)std::ceil(std::log2(v.size() / 2));
    const int threadsPerBlock = std::min(m_max_threads_per_block, i);
    const int blocksPerGrid = (i + threadsPerBlock - 1) / threadsPerBlock;
    async::kernel<<<blocksPerGrid, threadsPerBlock>>>(v, f, h2);
    async::boundaries<<<1, 1>>>(v, f, bcs, h2);
}
