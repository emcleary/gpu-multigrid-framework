#include "hip/hip_runtime.h"
#include "iterator.cuh"

#include <hip/hip_runtime.h>

#include "src/array.hpp"

using gmf::Array;
using gmf::ArrayRaw;
using gmf::Grid;
using gmf::modules::BoundaryConditions;


__host__ __device__
inline double eval(const ArrayRaw& v, const ArrayRaw& f, const double h, const double gamma, const int i) {
    double num = 2 * (h * h * f[i] + v[i+1] + v[i-1]);
    double denom = 4 + h * gamma * (v[i+1] - v[i-1]);
    return num / denom;
}

namespace iterator {
__global__
void kernel(ArrayRaw v, const ArrayRaw f, const double h, const double gamma, const int color) {
    int idx = 2 * (threadIdx.x + blockDim.x * blockIdx.x) + color;

    // must skip boundaries for Dirichlet BCs
    if (0 < idx && idx < v.size() - 1)
        v[idx] = eval(v, f, h, gamma, idx);
}

__global__
void boundaries(ArrayRaw v, const ArrayRaw f, const double gamma, const BoundaryConditions bcs, const double h) {
    const int n = v.size() - 1;
    
    if (bcs.is_left_dirichlet()) {
        v[0] = f[0];
    } else { // neumann
        double num = h * h * f[0] + 2 * v[1];
        double denom = 2 + gamma * bcs.get_left() * h * h;
        v[0] = num / denom;
    }

    if (bcs.is_right_dirichlet()) {
        v[n] = f[n];
    } else { // neumann
        double num = h * h * f[n] + 2 * v[n-1];
        double denom = 2 + gamma * bcs.get_right() * h * h;
        v[n] = num / denom;
    }
}
} // namespace iterator


void IteratorNonlinear::run_host(Array& v, const Array& f, const BoundaryConditions& bcs, const Grid& grid) {
    const int n = v.size() - 1;
    const double h = grid.get_cell_width();
    const double gamma = m_eqn->get_gamma();

    // Red-Black Gauss Seidel -- parallel and converges faster
    for (int i = 1; i < n; i += 2)
        v[i] = eval(v, f, h, gamma, i);
    for (int i = 2; i < n; i += 2)
        v[i] = eval(v, f, h, gamma, i);

    if (bcs.is_left_dirichlet()) {
        v[0] = f[0];
    } else { // neumann
        double num = h * h * f[0] + 2 * v[1];
        double denom = 2 + gamma * bcs.get_left() * h * h;
        v[0] = num / denom;
    }

    if (bcs.is_right_dirichlet()) {
        v[n] = f[n];
    } else { // neumann
        double num = h * h * f[n] + 2 * v[n-1];
        double denom = 2 + gamma * bcs.get_right() * h * h;
        v[n] = num / denom;
    }
}

void IteratorNonlinear::run_device(Array& v, const Array& f, const BoundaryConditions& bcs, const Grid& grid) {
    const int threadsPerBlock = std::min(m_max_threads_per_block, v.size() / 2);
    const int blocksPerGrid = (v.size() / 2 + threadsPerBlock - 1) / threadsPerBlock;
    const double h = grid.get_cell_width();
    const double gamma = m_eqn->get_gamma();

    // Red-Black Gauss Seidel
    iterator::kernel<<<blocksPerGrid, threadsPerBlock>>>(v, f, h, gamma, 1);
    iterator::kernel<<<blocksPerGrid, threadsPerBlock>>>(v, f, h, gamma, 2);
    iterator::boundaries<<<1, 1>>>(v, f, gamma, bcs, h);
}
