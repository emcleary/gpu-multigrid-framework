#include "hip/hip_runtime.h"
#include "iterator_async.cuh"

#include <hip/hip_runtime.h>

#include "src/array.hpp"

using pmf::Array;
using pmf::ArrayRaw;
using pmf::Grid;
using pmf::modules::BoundaryConditions;


namespace async {
__global__
void kernel(ArrayRaw v, const ArrayRaw f, const double h2) {
    int i = 2 * (threadIdx.x + blockDim.x * blockIdx.x);

    if (i + 1 < v.size() - 1)
        v[i+1] = (v[i] + v[i+2] + h2 * f[i+1]) / 2;

    if (0 < i && i < v.size() - 1)
        v[i] = (v[i-1] + v[i+1] + h2 * f[i]) / 2;
}

__global__
void boundaries(ArrayRaw v, const ArrayRaw f, const BoundaryConditions bcs, const double h2) {
    const int nh = v.size() - 1;

    if (bcs.is_periodic_x()) {
        v[0] = (v[nh-1] + v[1] + f[0] * h2) / 2;
        v[nh] = v[0];
    } else {
        if (bcs.is_west_dirichlet()) {
            v[0] = f[0];
        } else { // neumann
            v[0] = v[1] + f[0] * h2 / 2;
        }

        if (bcs.is_east_dirichlet()) {
            v[nh] = f[nh];
        } else { // neumann
            v[nh] = v[nh-1] + f[nh] * h2 / 2;
        }
    }
}
} // namespace async


void IteratorAsync::run_device(Array& v, const Array& f, const BoundaryConditions& bcs, const Grid& grid) {
    const double h2 = grid.get_cell_width() * grid.get_cell_width();
    const uint n = v.size() / 2;
    const uint threads = std::min(m_max_threads_per_block, n);
    const uint blocks = (n + threads - 1) / threads;
    async::kernel<<<blocks, threads>>>(v, f, h2);
    hipDeviceSynchronize();
    async::boundaries<<<1, 1>>>(v, f, bcs, h2);
}
